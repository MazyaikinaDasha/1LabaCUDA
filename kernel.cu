#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include ""

#define BLOCK_SIZE 32

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);


__global__ void Mult(float* a, float* b, int n, float* c)
{
	// ����� �����
	int bx = blockIdx.x; 
	int by = blockIdx.y; 
	// ����� ���� 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 

	float sum = 0.0f;
	// ����� ������ �� A
	int ia = n * (BLOCK_SIZE * by + ty); 
	// ����� ������� �� B
	int ib = BLOCK_SIZE * bx + tx; 
	// ����� �������� �� �
	int ic = ia + ib; 
	for (int k = 0; k < n; k++) {
		sum += a[ia + k] * b[ib + k * n];
	}
	c[ic] = sum;
}

int main()
{
	int N = 2048;
	int m, n, k;
	// �������� ����������-�������
	float timerValueGPU, timerValueCPU = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	int numBytes = N * N * sizeof(float);
	float* adev, * bdev, * cdev, * a, * b, * c, * cc;
	//������� A
	a = (float*)malloc(numBytes); 
	//������� B
	b = (float*)malloc(numBytes); 
	//������� � ��� GPU-��������
	c = (float*)malloc(numBytes); 
	//������� � ��� CPU-��������
	cc = (float*)malloc(numBytes); 

	// ������� ������� A, B � ����������������� ������� B
	for (n = 0; n < N; n++)
	{
		for (m = 0; m < N; m++)
		{
			a[m + n * N] = 2.0f * m + n; b[m + n * N] = m - n;
		}
	}
	// ������� ����� ����� � ������
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);
	// ��������� ������ �� GPU
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);


	// GPU-������� 
	// ����������� ������ A � B � host �� device
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	// ������ �������-����
	Mult << < blocks, threads >> > (adev, bdev, N, cdev);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time %f msec\n", timerValueGPU);
	// �����������, ����������� ������� C � device �� host
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	//  CPU-������� 
	double start_time = clock();
	// ���������� ������� C
	for (int i = 0; i < N; i++) {
		for (int k = 0; k < N; k++) {
			for (int j = 0; j < N; j++) {
				cc[i * N + j] += a[i * N + k] * b[k * N + j];
			}
		}
	}
	double end_time = clock();
	timerValueCPU = ((end_time - start_time)) * 1000 / CLOCKS_PER_SEC;

	printf("\n CPU calculation time %f msec\n", timerValueCPU);
	printf("\n Rate %f x\n", timerValueCPU / timerValueGPU);

	// ������������ ������ �� GPU � CPU
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);
	free(a);
	free(b);
	free(c);
	free(cc);
	// ����������� ����������-�������
	hipEventDestroy(start);
	hipEventDestroy(stop);
	system("pause");
	return 0;

}